#include <hip/hip_runtime.h>
#include "simulation.h"

namespace Cuda
{

__host__
void store(Particle* &dev, Particle const *particles, size_t size)
{
    hipMalloc(&dev, size);
    hipMemcpy(dev, particles, size, hipMemcpyHostToDevice);
}

__host__
void read_out(Particle *dev, Particle* &res, size_t size)
{
    hipMemcpy(res, dev, size, hipMemcpyDeviceToHost);
}

}
