#include "hip/hip_runtime.h"
#include "prediction_step.cuh"

namespace Cuda
{

__global__
void prediction_step(Particle *particles, float dt)
{
    Particle &p = particles[threadIdx.x];
    const unsigned int i = threadIdx.y;

    p.pos[i] += p.vel[i] * dt + 0.5 * dt * dt * p.F[i];
}

__host__
void run_prediction_step(int N, Particle *d_particles, float dt)
{
    prediction_step <<< 1, dim3(N, 3) >>>
        (d_particles, dt);
}

}
